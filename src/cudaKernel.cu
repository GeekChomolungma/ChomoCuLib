#include <hip/hip_runtime.h>

namespace cudaCal {

__global__ void addKernel(const int *a, const int *b, int *c, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

void calculateKernel(const int *a, const int *b, int *c, int size) {
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

} // namespace cudaCal